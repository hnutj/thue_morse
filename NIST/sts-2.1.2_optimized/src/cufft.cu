#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <>
#include "../include/interface.h"
#include "../include/common.cuh"

hipError_t ErrorCheck(hipError_t error_code, const char* filename, int lineNumber)
{
    if (error_code != hipSuccess)
    {
        printf("CUDA error:\r\ncode=%d, name=%s, description=%s\r\nfile=%s, line%d\r\n",
                error_code, hipGetErrorName(error_code), hipGetErrorString(error_code), filename, lineNumber);
        return error_code;
    }
    return error_code;
}

void cufft(int n, double* input, double* output){

    hipSetDevice(0);
    int i;
    hipfftDoubleReal *idata_cpu;
    hipfftDoubleComplex *odata_cpu;
    hipfftDoubleReal *idata_device;
    hipfftDoubleComplex *odata_device;
    
    idata_cpu=(hipfftDoubleReal*)malloc(n*sizeof(hipfftDoubleReal));
    odata_cpu=(hipfftDoubleComplex*)malloc((n/2+1)*sizeof(hipfftDoubleComplex));
    for(i=0;i<n;i++)
    {
        idata_cpu[i]=input[i];
    }
    
    // host to device
    // ErrorCheck(hipFree(0),__FILE__, __LINE__);
    ErrorCheck(hipMalloc((void**)&idata_device,n*sizeof(hipfftDoubleReal)),__FILE__, __LINE__);
    ErrorCheck(hipMalloc((void**)&odata_device,(n/2+1)*sizeof(hipfftDoubleComplex)),__FILE__, __LINE__);

    // 使用流化还可以再提高效率
    ErrorCheck(hipMemcpy(idata_device,idata_cpu,n*sizeof(hipfftDoubleReal),hipMemcpyHostToDevice),__FILE__, __LINE__);

    // exec fft     
    // 单精度使用接口hipfftExecR2C，双精度为D2Z
    hipfftHandle plan;
    hipfftPlan1d(&plan,n,HIPFFT_D2Z,1);
    hipfftExecD2Z(plan,(hipfftDoubleReal*)idata_device,(hipfftDoubleComplex*)odata_device);
    hipDeviceSynchronize();
    
    // device to host
    ErrorCheck(hipMemcpy(odata_cpu,odata_device,(n/2+1)*sizeof(hipfftDoubleComplex),hipMemcpyDeviceToHost),__FILE__, __LINE__);

    // for(i=0;i<n/2+1;i++)
    // {
    //     printf("%lf",odata_cpu[i].x);
    //     if(odata_cpu[i].y != 0.0 )
    //     {
    //         printf("+%lfi",odata_cpu[i].y);
    //     } 
    //     printf("\n");
    // }

	output[0] = sqrt(odata_cpu[0].x * odata_cpu[0].x);	    /* COMPUTE MAGNITUDE */
	
    //这边也可以用并行，速度会更快
	for ( i=1; i<n/2+1; i++ )
		output[i] = sqrt(pow(odata_cpu[i].x,2)+pow(odata_cpu[i].y,2)); 

    // for(i=0;i<n/2+1;i++)
    // {
    //     printf("%lf ",output[i]);
    // }
    // printf("\n");

    hipfftDestroy(plan);
    
    free(idata_cpu);
    free(odata_cpu);
    hipFree(odata_device);
    hipFree(idata_device);
}