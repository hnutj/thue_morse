#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "../include/interface.h"
#include "../include/common.cuh"

/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * *
          N O N O V E R L A P P I N G  T E M P L A T E  T E S T
 * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

// 手工实现(htonl(((int*)(ByteEpsilon+i))[0]
__device__ unsigned int myHtonls(unsigned int num1, unsigned int num2, int numOff1) 
{
	unsigned int result = 0;
	int numOff2 = numOff1 - 1;
	num1 >>= 8 * numOff1;
	for (int i = numOff1; i <= 3; i++) {
		result = result << 8 | (num1 & 0b11111111);
		num1 >>= 8;
	}
	for (int i = numOff2; i >= 0; i--) {
		result = result << 8 | (num2 & 0b11111111);
		num2 >>= 8;
	}
	return result;
}

//核函数
__global__ void NonOverlapping(int m, int n,unsigned char *cudaByteEpsilon,unsigned short *cudeTemplates, unsigned int *cudaContainer)
{
    const int jj = threadIdx.x;
    const int j = blockIdx.x;

	int N = 8;
	int M = n/N;
    unsigned short mask = powf(2, m) - 1;
    unsigned short B = cudeTemplates[jj]; //置于内层循环内外均可,置于内层为的是折叠循环便于omp并行
    //每个子块的索引区域：j*blockSize~(j+1)*blockSize-m
    int beginPos = j * M, endPos = beginPos + M - m;
    int beginByte = beginPos / 8, beginOff = beginPos % 8;
    int endByte = endPos / 8, endOff = endPos % 8;

    int W_obs = 0; //每个模板下每个子块有一个W_obs

    int p = beginByte, t=beginOff;
    while (p <= endByte) {
        int pos2 = (p == endByte ? endOff : 7);
        if( t > pos2){
            break;
        }
        while (t <= pos2) {
            unsigned short test;
            int k = 8 - t - m;
            if (k >= 0) {
                // 涉及1byte
                test = (cudaByteEpsilon[p] >> k) & mask;
            }
            else if (k >= -8) {
                //涉及2bytes
                test = ((cudaByteEpsilon[p] << (-k)) | (cudaByteEpsilon[p + 1] >> (8 + k))) & mask;
            }
            else if (k >= -15) {
                //涉及3bytes
				int numOff1 = p % 4;
				int num1 = ((int*)(cudaByteEpsilon + p / 4 * 4))[0];
				int num2 = ((int*)(cudaByteEpsilon + (p / 4 + 1) * 4))[0];
				test = (myHtonls(num1, num2, numOff1) >> (24 + k)) & mask;			
            }
            if (test == B) {
                //match
                W_obs++;
                //int nextPos = p * 8 + t + m;
                p += (t + m) / 8;
                t = (t + m) % 8;
                break;
            }
            else if(t++ == pos2){
                t = 0;
                p++;
                break;
            }
        }
    }
    cudaContainer[jj*N+j] = W_obs;    
}

void NonOverlappingHelper(int m, int n, int templateNum, unsigned char* byteEpsilon, unsigned short* templates, unsigned int*container)
{
	// 锁定分块数为8
	int N = 8;
	// 每个子块的长度
	// int M = n/N;

    // 1、设置GPU设备
    // setGPU();
    hipSetDevice(0);
    
    // 2、分配主机内存和设备内存，并初始化
    int byteNum=n / 8;
    int byteTotalNum = n % 8 == 0 ? byteNum + 4 : byteNum + 5;
    int stBytesCount1 = byteTotalNum * sizeof(unsigned char);
    int stBytesCount2 = templateNum * sizeof(unsigned short);
    int stBytesCount3 = templateNum * N * sizeof(unsigned int);

    unsigned char *cudaByteEpsilon;
    unsigned short *cudeTemplates;
    unsigned int *cudaContainer;

    //一次初始化后，程序cudamalloc()分配的内存不释放，继续使用，所有程序运行结束后，再一起释放
    ErrorCheck(hipFree(0),__FILE__, __LINE__);

    ErrorCheck(hipMalloc((void**)&cudaByteEpsilon, stBytesCount1),__FILE__, __LINE__);
    ErrorCheck(hipMalloc((void**)&cudeTemplates, stBytesCount2),__FILE__, __LINE__);
    ErrorCheck(hipMalloc((void**)&cudaContainer, stBytesCount3),__FILE__, __LINE__);

    // if (cudaByteEpsilon != NULL && cudeTemplates != NULL && cudaContainer != NULL)
    // {
    //     ErrorCheck(hipMemset(cudaByteEpsilon, 0,stBytesCount1),__FILE__, __LINE__);  // 设备内存初始化为0
    //     ErrorCheck(hipMemset(cudeTemplates, 0, stBytesCount2),__FILE__, __LINE__);
    //     ErrorCheck(hipMemset(cudaContainer, 0, stBytesCount3),__FILE__, __LINE__);
    // }
    // else
    // {
    //     printf("fail to allocate memory\n");
    //     free(cudaByteEpsilon);
    //     free(cudeTemplates);
    //     free(cudaContainer);
    //     exit(-1);
    // }

    // 3、数据从主机复制到设备
    ErrorCheck(hipMemcpy(cudaByteEpsilon, byteEpsilon, stBytesCount1, hipMemcpyHostToDevice),__FILE__, __LINE__); 
    ErrorCheck(hipMemcpy(cudeTemplates, templates, stBytesCount2, hipMemcpyHostToDevice),__FILE__, __LINE__); 

    // 4、调用核函数
    dim3 block(templateNum);
    dim3 grid(N);
    NonOverlapping<<<grid, block>>>(m, n, cudaByteEpsilon, cudeTemplates, cudaContainer);   
    ErrorCheck(hipGetLastError(), __FILE__, __LINE__);
    ErrorCheck(hipDeviceSynchronize(), __FILE__, __LINE__);

    // 5、将计算得到的数据从设备传给主机
    ErrorCheck(hipMemcpy(container, cudaContainer, stBytesCount3, hipMemcpyDeviceToHost),__FILE__, __LINE__);

    // 6、释放内存
    // ErrorCheck(hipFree(cudaByteEpsilon),__FILE__, __LINE__);
    // ErrorCheck(hipFree(cudeTemplates),__FILE__, __LINE__);
    // ErrorCheck(hipFree(cudaContainer),__FILE__, __LINE__);

    // ErrorCheck(hipDeviceReset(),__FILE__, __LINE__);	
}
